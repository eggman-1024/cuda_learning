#include "hip/hip_runtime.h"
#include "../include/add2.h"

__global__ void MatAdd(float* c, 
                      const float* a, 
                      const float* b, 
                      int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = j * n + i;
    if (i < n && j < n) {
        c[idx] = a[idx] + b[idx];
    }
}

void launch_add2(float* c, 
                 const float* a, 
                 const float* b, 
                 int n) {
    dim3 block(16, 16); // 定义每个block有16×16=256个线程
    dim3 grid(n/block.x, n/block.y);    // 定义每个block有16×16=256个线程
    MatAdd<<<grid, block>>>(c, a, b, n);    // 启动CUDA kernel
}